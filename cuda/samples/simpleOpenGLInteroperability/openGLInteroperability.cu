#define GL_GLEXT_PROTOTYPES
#include <GL/glut.h>

#include<iostream>

#include<cuda_gl_interop.h>
#include<hip/hip_runtime_api.h>

#define DIM 512

GLuint bufferObj;
hipGraphicsResource *resource;


int
main(int argc, char *argv[]){
  hipDeviceProp_t prop;
  int dev;
  memset(&prop,0, sizeof(hipDeviceProp_t));
  prop.major = 1;
  prop.minor = 0;
  checkCudaErrors(hipChooseDevice(&dev, &prop));

  checkCudaErrors(cudaGLSetGLDevice(dev));

  glutInit(&argc, argv);
  glutInitDisplayMode(GLUT_DOUBLE|GLUT_RGBA);
  glutInitWindowSize(DIM,DIM);
  glutCreateWindow("bitmap");
}


