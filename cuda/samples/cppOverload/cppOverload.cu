#include "hip/hip_runtime.h"
#include<iostream>
#include<memory>

#include<hip/hip_runtime_api.h>

using namespace std;

#define N 1024
#define NUM_THREADS 256
#define DIV_UP(a,b) (((a)+(b)-1)/(b))

#include"kernel_overload.cuh"

typedef void(*fp)(int*);

int
_main(int argc, char const *argv[]){

  cout<<argv[0]<<" Starting..."<<endl;

  int devID = 0;
  hipDeviceProp_t prop;
  checkCudaErrors(hipGetDeviceProperties(&prop,devID));
  if(prop.major<2){
    cerr<<"ERROR: cppOverload requireds sm 2.0 or higher"<<endl;
    cout<<"Current GPU device has SM "<<prop.major<<"."<<prop.minor<<" Exiting..."<<endl;
    exit(EXIT_FAILURE);
  }
  checkCudaErrors(hipSetDevice(devID));

  //分配内存
  int *_dInput;
  int *_dOutput;
  int *_hInput;
  int *_hOutput;
  checkCudaErrors(hipMalloc((void**)&_dInput, 2*N*sizeof(int) ) );
  checkCudaErrors(hipMalloc(&_dOutput, sizeof(int)*N));
  unique_ptr<int,fp>dInput{_dInput, [](int*p){hipFree(p);} };
  unique_ptr<int,fp>dOutput{_dOutput, [](int*p){hipFree(p);}};

  checkCudaErrors(hipHostMalloc(&_hInput, sizeof(int)*N*2));
  checkCudaErrors(hipHostMalloc(&_hOutput, sizeof(int)*N));
  unique_ptr<int,fp>hInput{_hInput, [](int*p){hipHostFree(p);}};
  unique_ptr<int,fp>hOutput{_hOutput, [](int*p){hipHostFree(p);}};

  for(int i=0;i<N*2; i++){
    *(hInput.get()+i) = i;
  }
  checkCudaErrors(hipMemcpy(dInput.get(),hInput.get(),
                             sizeof(int)*N*2, hipMemcpyHostToDevice ));

  //测试cpp重载
  int a = 1;
  
  simple_kernel<<< DIV_UP(N,NUM_THREADS), NUM_THREADS>>>(dInput.get(), dOutput.get(), a);
  checkCudaErrors(hipMemcpy(hOutput.get(), dOutput.get(), sizeof(int)*N, hipMemcpyDeviceToHost));

  simple_kernel<<<DIV_UP(N,NUM_THREADS),NUM_THREADS>>>(reinterpret_cast<int2*>(dInput.get()), dOutput.get(),a);
  checkCudaErrors(hipMemcpy(hOutput.get(), dOutput.get(), sizeof(int)*N, hipMemcpyDeviceToHost));

  simple_kernel<<<DIV_UP(N,NUM_THREADS),NUM_THREADS>>>(dInput.get(),dInput.get()+N, dOutput.get(),a);
  checkCudaErrors(hipMemcpy(hOutput.get(), dOutput.get(), sizeof(int)*N, hipMemcpyDeviceToHost));

  checkCudaErrors(hipDeviceSynchronize());
//  hipFree(_dOutput);
//  hipHostFree(_hInput);
//  hipHostFree(_hOutput);
  return 0;
}

int 
main(int argc, char const*argv[]){
  _main(argc, argv);
  checkCudaErrors(hipDeviceReset());
  
}
