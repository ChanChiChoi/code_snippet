#include "hip/hip_runtime.h"
#include<iostream>

using namespace std;

//#include<hip/hip_runtime.h>

#include<hip/hip_runtime_api.h>
//#include<helper_functions.h>

__global__ void
increment_kernel(int *g_data, int inc_value){

  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  g_data[tid] = g_data[tid] + inc_value;

}

bool corrent_output(int *data, int const n, int const x){

  for(int i=0; i<n; i++){
    if(data[i] != x){
       cout<<"Error! data["<<i<<"] = "<<data[i]
           <<", ref = "<<x<<endl;
    }
  }
  return true;
}

int main(int argc, char *argv[]){

  int devID;
  hipDeviceProp_t deviceProps;

  cout<<argv[0]<<" - Starting..."<<endl;
  
  char const * tmp1 = *argv;
  char const ** tmp2 =  &tmp1;
  devID = findCudaDevice(argc,tmp2);

  checkCudaErrors(hipGetDeviceProperties(&deviceProps, devID));
  cout<<"CUDA device: "<<deviceProps.name<<endl;

  //===========================
  int n = 16*1024*1024;
  int nbytes = n*sizeof(int);
  int value = 26;
  
  // host memory
  int *a = 0;
  //checkCudaErrors(hipHostMalloc((void **)&a, nbytes));

  checkCudaErrors(hipHostAlloc( (void **)&a, nbytes, hipHostMallocDefault));
  memset(a, 0, nbytes);


  // device memory
  int *d_a = 0;
  checkCudaErrors(hipMalloc((void **)&d_a, nbytes));
  checkCudaErrors(hipMemset(d_a, 255, nbytes));

  // set kernel launch configuration
  dim3 threads = 512;
  dim3 blocks = n / threads.x;

  // create cuda event handles
  hipEvent_t start, stop;
  checkCudaErrors(hipEventCreate(&start));
  checkCudaErrors(hipEventCreate(&stop));

  checkCudaErrors(hipDeviceSynchronize());
  float gpu_time = 0.0f;

  //asnychronously issue work to the gpu(all to stream 0)
  hipEventRecord(start,0);
  hipMemcpyAsync(d_a, a, nbytes, hipMemcpyHostToDevice, 0);
  increment_kernel<<<blocks, threads, 0, 0>>>(d_a, value);
  hipMemcpyAsync(a, d_a, nbytes, hipMemcpyDeviceToHost, 0);
  hipEventRecord(stop, 0);

  // have cpu do some work while waiting for stage 1 to finish
  unsigned long int counter=0;
  
  while(hipEventQuery(stop) == hipErrorNotReady)
    counter++;

  checkCudaErrors(hipEventElapsedTime(&gpu_time, start, stop));
  
  // print cpu and gpu times
  cout<<"time spent executing by the gpu: "<<gpu_time<<endl;
  cout<<"cpu execute "<< counter<<" iterations while waiting for gpu to finish"<<endl;

  // check the output for correntness
  bool bFinalRes = corrent_output(a, n, value);

  // release resources
  checkCudaErrors(hipEventDestroy(start));
  checkCudaErrors(hipEventDestroy(stop));
  checkCudaErrors(hipHostFree(a));
  checkCudaErrors(hipFree(d_a));

  exit(bFinalRes ? EXIT_SUCCESS: EXIT_FAILURE);

}
