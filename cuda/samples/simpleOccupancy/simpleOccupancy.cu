#include "hip/hip_runtime.h"
#include<iostream>
#include<hip/hip_runtime_api.h>

using std::cout;
using std::endl;
using std::cerr;

constexpr int manualBlockSize = 32;

__global__ void
square(int *array, int arrayCount){

  extern __shared__ int dynamicSMem[];
  int idx = threadIdx.x + blockIdx.x*blockDim.x;
  if(idx < arrayCount)
    array[idx] *= array[idx];
}


double
reportPotentialOccupancy(void *kernel, int block, size_t dynamicSMem){
  int device;
  hipDeviceProp_t prop;
  int nBlock;
  int activeWarps;
  int maxWarps;
  double occupancy;
  checkCudaErrors(hipGetDevice(&device));
  checkCudaErrors(hipGetDeviceProperties(&prop,device));
  checkCudaErrors(hipOccupancyMaxActiveBlocksPerMultiprocessor(
                      &nBlock,
                      kernel,
                      block,
                      dynamicSMem
                     ));

  activeWarps = nBlock*block / prop.warpSize;
  maxWarps = prop.maxThreadsPerMultiProcessor / prop.warpSize;
  occupancy = (double)activeWarps / maxWarps;
  return occupancy;
}

int
launchConfig(int *array, int arrayCount, bool automatic){
  int block;
  int minGrid;
  int grid;
  size_t dynamicSMemUsage = 0;
  
  hipEvent_t st;
  hipEvent_t ed;
  float elapsedTime;
  double potentialOccupancy;
  checkCudaErrors(hipEventCreate(&st));
  checkCudaErrors(hipEventCreate(&ed));

  if(automatic){
    //Returns grid and block size that achieves maximum potential occupancy for a device function.
    checkCudaErrors(hipOccupancyMaxPotentialBlockSize(
                     &minGrid,
                     &block,
                     (void*)square,
                     dynamicSMemUsage,
                     arrayCount
                    ));
    cout<<"suggested block size: "<<block<<endl;
    cout<<"minimum grid size for maximum occupancy: "<<minGrid<<endl;
    cout<<"dynamic mem:"<<dynamicSMemUsage<<endl;
  }else{
    block = manualBlockSize;
  }
  grid = (arrayCount+ block-1)/block;
  checkCudaErrors(hipEventRecord(st));
  square<<<grid,block,dynamicSMemUsage>>>(array, arrayCount);
  checkCudaErrors(hipEventRecord(ed));
  checkCudaErrors(hipDeviceSynchronize());

  potentialOccupancy = reportPotentialOccupancy((void*)square, block,dynamicSMemUsage);
  cout<<"Potential occupancy: "<<potentialOccupancy*100<<"%"<<endl;
  checkCudaErrors(hipEventElapsedTime(&elapsedTime, st, ed));
  cout<<"Elapsed time: "<<elapsedTime<<" ms"<<endl;
  return 0;
  
  
}

int
test(bool automaticLaunchConfig, int const count = 1000000){
  int *array;
  int *dArray;
  int size = count*sizeof(int);
  array = new int[count];

  for(int i=0; i<count; i++)
    array[i] = i;

  checkCudaErrors(hipMalloc(&dArray, size));
  checkCudaErrors(hipMemcpy(dArray,array,size,hipMemcpyHostToDevice));
  for(int i=0; i<count; i++)
    array[i]=0;

  launchConfig(dArray, count, automaticLaunchConfig);

  checkCudaErrors(hipMemcpy(array, dArray, size, hipMemcpyDeviceToHost));
  checkCudaErrors(hipFree(dArray));

  for(int i=0; i<count; i++)
    if(array[i] != i*i){
      cout<<"element:"<<i<<" expected:"<<i*i<<" actual:"<<array[i]<<endl;
      return 1;
    }

  delete [] array;
  return 0;
}


int
main(){
  int status;
  //----------
  cout<<"starting simple occupancy"<<endl<<endl;
  cout<<"[ manual configuration with "<<manualBlockSize
      <<" threads per block ]"<<endl;
  status = test(false);
  if(status){
    cerr<<"Test Failed"<<endl;
    return -1;
  }
  //-----------
  cout<<endl;
  cout<<"[ Automic, occupancy-based configuration ]"<<endl;
  status = test(true);
  if(status){
    cerr<<"Test Failed"<<endl;
    return -1;
  }
  //----------
  cout<<endl;
  cout<<"Test PASSED"<<endl;
  return 0;
  
}
