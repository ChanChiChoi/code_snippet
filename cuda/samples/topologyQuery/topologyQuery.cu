#include<iostream>
#include<hip/hip_runtime_api.h>

using std::cout;
using std::endl;

int
main(int argc, char *argv[]){
  int deviceCount = 0;
  checkCudaErrors(hipGetDeviceCount(&deviceCount));

  for(int device1 = 0; device1<deviceCount; device1++){

    for(int device2 = 0; device2<deviceCount; device2++){
      if(device1 == device2)
        continue;
      int perfRank=0;
      int atomicSupported = 0;
      int accessSupported = 0;

      checkCudaErrors(hipDeviceGetP2PAttribute(&accessSupported,
                            hipDevP2PAttrAccessSupported, device1, device2));
      checkCudaErrors(hipDeviceGetP2PAttribute(&perfRank,
                            hipDevP2PAttrPerformanceRank, device1, device2));
      checkCudaErrors(hipDeviceGetP2PAttribute(&atomicSupported,
                            hipDevP2PAttrNativeAtomicSupported, device1, device2));

      if(accessSupported){
         cout<<"GPU"<<device1<<" <-> GPU"<<device2<<":"<<endl;
         cout<<" * Atomic Supported: "<<(atomicSupported?"Yes":"No")<<endl;
         cout<<" * Perf Rank: "<<perfRank<<endl;
      }
    }
  }
  for(int device = 0; device<deviceCount; device++){
    
      int atomicSupported = 0;
      checkCudaErrors(hipDeviceGetAttribute(&atomicSupported,
                            hipDeviceAttributeHostNativeAtomicSupported, device));
      cout<<"GPU"<<device<<" <-> CPU: "<<endl;
      cout<<" * Atomic Supported: "<<(atomicSupported?"Yes":"No")<<endl;
  }

}
