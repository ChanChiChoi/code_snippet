#include "hip/hip_runtime.h"
#include<cstdio>
#include<cstdlib>
#include<iostream>
#include<ctime>
#include<vector>
#include<algorithm>

#ifdef USE_PTHREADS
#include<pthread.h>
#else
#include <omp.h>
#endif

#include<hipblas.h>
#include<hip/hip_runtime_api.h>

using std::vector;
using std::cout;
using std::endl;


template<typename T>
struct Task{
  unsigned int size, id;
  T *data;
  T *result;
  T *vector;

  Task():size(0), id(0), data(nullptr), 
         result(nullptr), vector(nullptr){}
  Task(unsigned int s):size(s), id(0), data(nullptr),
                       result(nullptr){
    checkCudaErrors(hipMallocManaged(&data, sizeof(T)*size*size));
    checkCudaErrors(hipMallocManaged(&result, sizeof(T)*size));
    checkCudaErrors(hipMallocManaged(&vector, sizeof(T)*size));
    checkCudaErrors(hipDeviceSynchronize());
  }

  ~Task(){
    checkCudaErrors(hipDeviceSynchronize());
    checkCudaErrors(hipFree(data));
    checkCudaErrors(hipFree(result));
    checkCudaErrors(hipFree(vector));
  }

  void allocate(unsigned int const s, unsigned int const unique_id){
    id = unique_id;
    size = s;
    checkCudaErrors(hipMallocManaged(&data, sizeof(T)*size*size));
    checkCudaErrors(hipMallocManaged(&result, sizeof(T)*size));
    checkCudaErrors(hipMallocManaged(&vector, sizeof(T)*size));
    checkCudaErrors(hipDeviceSynchronize());

    for(int i=0; i<size*size; i++)
      data[i] = (T)((float)rand()/RAND_MAX*100);

    for(int i=0; i<size; i++){
      result[i] = 0;
      vector[i] = (T)((float)rand()/RAND_MAX*100);
    }
  }

};

#ifdef USE_PTHREADS
typedef struct threadData_t{
  int tid;
  hipStream_t *streams;
  hipblasHandle_t *handles;
  int taskSize;

  Task<double> *TaskListPtr;
}threadData;
#endif

template<typename T> void 
initialize_tasks(vector<Task<T>> &TaskList){
  for(unsigned int i=0; i<TaskList.size(); i++){
    int size;
    size = max(64, (int)(T)((float)rand()/RAND_MAX));
    TaskList[i].allocate(size,i);
  }
}


template<typename T> void
gemv(int m, int n, T alpha, T *A, T *x, T beta, T *result){
  for(int i=0; i<m; i++){
    result[i] *= beta;
    for(int j=0; j<n; j++)
      result[i] += A[i*n+j]*x[j]; 
  }
}

#ifdef USE_PTHREADS
void execute(void *inpArgs){
  // 一个单独的线程，其中threadData中streams和handles都是整个数组
  threadData *dataPtr = (threadData*)inpArgs;
  hipStream_t *stream = dataPtr->streams;
  hipblasHandle_t *handle = dataPtr->handles;
  int tid = dataPtr->tid;

  for(int i=0; i<dataPtr->taskSize; i++){

    Task<double>&t = dataPtr->TaskListPtr[i];
    // 如果任务很小，就放到0号流中
    // 且用cpu操作，否则才用cublas
    // 一共4个线程，如果分到每个线程都小于100个任务，那么其实整体都在0号流中执行
    cout<<"Task ["<<t.id<<"], thread ["<<tid<<"] executing on host ("<<t.size<<")"<<endl;
    if(t.size < 100){
      // attach managed memory to a (dummy) stream to allow host access while the device is running
      checkCudaErrors(hipStreamAttachMemAsync(stream[0], t.data, 0, hipMemAttachHost));
      checkCudaErrors(hipStreamAttachMemAsync(stream[0], t.vector, 0, hipMemAttachHost));
      checkCudaErrors(hipStreamAttachMemAsync(stream[0], t.result, 0, hipMemAttachHost));
      // 确保异步hipStreamAttachMemAsync 已经执行了
      checkCudaErrors(hipStreamSynchronize(stream[0]));
      // call the host operation
      gemv(t.size, t.size, 1.0, t.data, t.vector, 0.0, t.result);
    }else{

      double one = 1.0;
      double zero = 0.0;
      checkCudaErrors(hipblasSetStream(handle[tid+1], stream[tid+1]));
      checkCudaErrors(hipStreamAttachMemAsync(stream[tid+1], t.data, 0, hipMemAttachSingle));
      checkCudaErrors(hipStreamAttachMemAsync(stream[tid+1], t.vector, 0, hipMemAttachSingle));
      checkCudaErrors(hipStreamAttachMemAsync(stream[tid+1], t.result, 0, hipMemAttachSingle));
      // 调用device的
      checkCudaErrors(hipblasDgemv(handle[tid+1], HIPBLAS_OP_N, t.size, t.size, &one, t.data, t.size,
                                  t.vector, 1, &zero, t.result,1));

    }
  }
}
#else
template<typename T> void
execute(Task<T> &t, hipblasHandle_t *handle, hipStream_t *stream, int tid){

  cout<<"Task ["<<t.id<<"], thread ["<<tid<<"] executing on host ("<<t.size<<")"<<endl;

  if(t.size<100){
    checkCudaErrors(hipStreamAttachMemAsync(stream[0], t.data, 0, hipMemAttachHost));
    checkCudaErrors(hipStreamAttachMemAsync(stream[0], t.vector, 0, hipMemAttachHost));
    checkCudaErrors(hipStreamAttachMemAsync(stream[0], t.result, 0, hipMemAttachHost));
    // 确保异步hipStreamAttachMemAsync 已经执行了
    checkCudaErrors(hipStreamSynchronize(stream[0]));
    // call the host operation
    gemv(t.size, t.size, 1.0, t.data, t.vector, 0.0, t.result);
  }else{
    double one = 1.0;
    double zero = 0.0;
    checkCudaErrors(hipblasSetStream(handle[tid+1], stream[tid+1]));
    checkCudaErrors(hipStreamAttachMemAsync(stream[tid+1], t.data, 0, hipMemAttachSingle));
    checkCudaErrors(hipStreamAttachMemAsync(stream[tid+1], t.vector, 0, hipMemAttachSingle));
    checkCudaErrors(hipStreamAttachMemAsync(stream[tid+1], t.result, 0, hipMemAttachSingle));
    // 调用device的
    checkCudaErrors(hipblasDgemv(handle[tid+1], HIPBLAS_OP_N, t.size, t.size, &one, t.data, t.size,
                                t.vector, 1, &zero, t.result,1));
  }
}




#endif

int
main(int argc, char *argv[]){
  hipDeviceProp_t prop;
  int dev_id = 0;
  checkCudaErrors(hipGetDeviceProperties(&prop, dev_id));

  if(!prop.managedMemory){
    fprintf(stderr, "Unified Memory not supported on this device\n");
    exit(EXIT_FAILURE);
  }
  if(prop.computeMode == hipComputeModeProhibited){
    fprintf(stderr, "This sample requires a device in either default or process exclusive mode");
    exit(EXIT_FAILURE);
  }

  int seed = time(NULL);
  srand((unsigned int)seed);
  int const nthreads = 4;
  hipStream_t *streams = new hipStream_t[nthreads+1]; // streams[]
  hipblasHandle_t *handles = new hipblasHandle_t[nthreads+1];// handles[]

  for(int i=0; i<nthreads+1; i++){
    checkCudaErrors(hipStreamCreate(&streams[i]));
    checkCudaErrors(hipblasCreate(&handles[i]));
  }

  unsigned int N = 40;
  vector<Task<double>> TaskList(N);
  initialize_tasks(TaskList);

  cout<<"Executing tasks on host / device"<<endl;

#ifdef USE_PTHREADS
  pthread_t threads[nthreads];
  threadData *InputToThreads = new threadData[nthreads];
  for(int i=0; i<nthreads; i++){
    checkCudaErrors(hipSetDevice(dev_id));
    InputToThreads[i].tid = i;
    // TODO
    InputToThreads[i].streams = streams; // 的确是整个数组传进去的
    InputToThreads[i].handles = handles;
    // 如果刚好分配完
    if((TaskList.size()/nthreads) == 0){
      InputToThreads[i].taskSize = (TaskList.size()/nthreads);
      InputToThreads[i].TaskListPtr = &TaskList[i*(TaskList.size()/nthreads)];
    }else{
      // 最后一个接收所有剩下的任务
      if(i == nthreads - 1){
        InputToThreads[i].taskSize = (TaskList.size() / nthreads) + (TaskList.size()%nthreads);
        InputToThreads[i].TaskListPtr = &TaskList[i*(TaskList.size()/nthreads)+(TaskList.size()%nthreads)]
      }else{
        InputToThreads[i].taskSize = (TaskList.size()/nthreads);
        InputToThreads[i].TaskListPtr = &TaskList[i*(TaskList.size()/nthreads)];
      }
    }
    pthread_create(&threads[i], NULL, &execute, &InputToThreads[i]);
  }

  for(int i=0; i<nthreads; i++)
    pthread_join(threads[i], NULL);
#else
  omp_set_num_threads(nthreads);
  #pragma omp parallel for schedule(dynamic)
  for(unsigned int i=0; i<TaskList.size(); i++){
    checkCudaErrors(hipSetDevice(dev_id));
    int tid = omp_get_thread_num();
    execute(TaskList[i], handles, streams, tid);
  }
#endif

   checkCudaErrors(hipDeviceSynchronize());
   for(int i=0; i<nthreads+1; i++){
      hipStreamDestroy(streams[i]);
      hipblasDestroy(handles[i]);
   }
   //free tasklist
   vector<Task<double>>().swap(TaskList);
   cout<<"All Done"<<endl;
  
}
