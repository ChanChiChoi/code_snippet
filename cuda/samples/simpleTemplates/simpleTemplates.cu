#include<cstdio>
#include<cassert>
#include<cstring>
#include<cmath>

#include<hip/hip_runtime_api.h>
#include<timer.h>

#include "sharedmem.cuh"


template<typename T> void
runTest(int argc, char *argv[], int len, int g_TotalFailres){
  
  unsigned int num_threads = len;
  unsigned int mem_size = sizeof(float)*num_threads;

  // 分配host内存并初始化
  T *h_idata = (T*)malloc(mem_size);
  for(unsigned int i=0; i<num_threads; i++){
    h_idata[i] = (T)i;
  }  

  //分配device的输入内存并复制
  T *d_idata;
  checkCudaErrors(hipMalloc((void**)&d_idata, mem_size));
  checkCudaErrors(hipMemcpy(d_idata, h_idata, mem_size, hipMemcpyHostToDevice));
  //分配device的输出内存
  T *d_odata;
  checkCudaErrors(hipMalloc((void**)&d_odata, mem_size));
  // kernel调度
  dim3 grid(1,1,1);
  dim3 threads(num_threads,1,1);

  testKernel<T><<<grid,threads,mem_size>>>(d_idata, d_odata);

  getLastCudaError("Kernel execution failed");
  // 分配host侧的结果内存,然后结果cp到host
  T *h_odata = (T*)malloc(mem_size); 
  checkCudaErrors(hipMemcpy(h_odata, d_odata, sizeof(T)*num_threads,
                             hipMemcpyDeviceToHost));
  
  
}


int
main(int argc, char *argv[]){

  int g_TotalFailres=0;
  cout<<"> runTest<float, 32>"<<endl;
  runTest<float>(argc, argv, 32,g_TotalFailres);
  cout<<"> runTest<int, 64>"<<endl;
  runTest<int>(argc, argv, 64, g_TotalFailres);

  cout<<"[simpleTemplates] -> Test Results: "<<g_TotalFailres<<" Failures"<<endl;
  exit(g_TotalFailres == 0? EXIT_SUCCESS: EXIT_FAILURE);
}

//需要使用模板特例化来解决针对不同类型下CUTIL的数组对比和文件写功能
