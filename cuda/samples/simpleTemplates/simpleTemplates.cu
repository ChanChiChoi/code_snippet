#include "hip/hip_runtime.h"
#include<cstdio>
#include<iostream>
#include<cassert>
#include<cstring>
#include<cmath>
#include<chrono>

#include<hip/hip_runtime_api.h>
#include<timer.h>

// 当在main中只有一次实例化runTest时，之前写法是没问题的
// 但是当实例化超过2回，编译器内部其实就需要创建两次定义，
//*****************************************************
// #include<cstdio>
// template<typename T>
// __global__ void
// test(){
//   extern __shared__ T data[];
//   printf("inside\n");
// }
// 
// int main(){
//   test<int><<<3,4,5>>>();
//   test<double><<<3,4,5>>>();
//   hipDeviceSynchronize();
// }
//*****************************************************
// 但是extern是外部的意思，从而会造成冲突
// 所以需要采用诸如sharedmem.cuh中的进行封装

#include "sharedmem.cuh"

using std::cout;
using std::endl;
using std::chrono::system_clock;
using std::chrono::milliseconds;
using std::chrono::duration_cast;

template<typename T> __global__ void
testKernel(T *g_idata, T *g_odata){

  SharedMemory<T> smem;
  T *sdata = smem.getPointer();
//  extern __shared__ T sdata[];

  unsigned int const tid = threadIdx.x;
  unsigned int const num_threads = blockDim.x;
  
  //global -> shared
  sdata[tid] = g_idata[tid];
  __syncthreads();
  
  //
  sdata[tid] = (T)num_threads * sdata[tid];
  __syncthreads();
  //shared -> global
  g_odata[tid] = sdata[tid];
}

template<typename T> void
runTest(int argc, char *argv[], int len, int g_TotalFailres){
  
  unsigned int num_threads = len;
  unsigned int mem_size = sizeof(float)*num_threads;

  auto st = system_clock::now();

  // 分配host内存并初始化
  T *h_idata = (T*)malloc(mem_size);
  for(unsigned int i=0; i<num_threads; i++){
    h_idata[i] = (T)i;
  }  

  //分配device的输入内存并复制
  T *d_idata;
  checkCudaErrors(hipMalloc((void**)&d_idata, mem_size));
  checkCudaErrors(hipMemcpy(d_idata, h_idata, mem_size, hipMemcpyHostToDevice));
  //分配device的输出内存
  T *d_odata;
  checkCudaErrors(hipMalloc((void**)&d_odata, mem_size));
  // kernel调度
  dim3 grid(1,1,1);
  dim3 threads(num_threads,1,1);

  testKernel<T><<<grid,threads,mem_size>>>(d_idata, d_odata);

  getLastCudaError("Kernel execution failed");
  // 分配host侧的结果内存,然后结果cp到host
  T *h_odata = (T*)malloc(mem_size); 
  checkCudaErrors(hipMemcpy(h_odata, d_odata, sizeof(T)*num_threads,
                                hipMemcpyDeviceToHost));
  auto ed = system_clock::now();
  cout<<"take "<<duration_cast<milliseconds>(ed-st).count()<<" ms"<<endl;

  T *ref = (T*)malloc(mem_size);
  
  
}


int
main(int argc, char *argv[]){

  int g_TotalFailres=0;
  cout<<"> runTest<float, 32>"<<endl;
  runTest<float>(argc, argv, 32,g_TotalFailres);
  cout<<"> runTest<int, 64>"<<endl;
  runTest<int>(argc, argv, 64, g_TotalFailres);

  cout<<"[simpleTemplates] -> Test Results: "<<g_TotalFailres<<" Failures"<<endl;
  exit(g_TotalFailres == 0? EXIT_SUCCESS: EXIT_FAILURE);
}

//需要使用模板特例化来解决针对不同类型下CUTIL的数组对比和文件写功能
